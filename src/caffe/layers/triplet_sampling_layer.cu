#include "hip/hip_runtime.h"
// Copyright 2014 BVLC and contributors.

#include <algorithm>
#include <cmath>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/util/io.hpp"

namespace caffe {

using std::max;


template <typename Dtype>
void TripletSamplingLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, vector<Blob<Dtype>*>* top) {
  Forward_cpu(bottom, top);
}

template <typename Dtype>
void TripletSamplingLayer<Dtype>::Backward_gpu(
    const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down,
    vector<Blob<Dtype>*>* bottom) {
  Backward_cpu(top, propagate_down, bottom);
}

INSTANTIATE_CLASS(TripletSamplingLayer);

}  // namespace caffe
