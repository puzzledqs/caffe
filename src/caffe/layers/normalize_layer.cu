#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "thrust/device_vector.h"

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

#define EPSILON 1e-6

namespace caffe {

template <typename Dtype>
void NormalizeLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = (*top)[0]->mutable_gpu_data();
  Dtype normsqr;
  int num = bottom[0]->num();
  int dim = bottom[0]->count() / num;
  for (int i = 0; i < num; ++i) {
    caffe_gpu_dot(dim, bottom_data + i*dim, bottom_data + i*dim, &normsqr);
    normsqr += EPSILON;
    caffe_gpu_scale<Dtype>(dim, pow(normsqr, -0.5), bottom_data + i*dim, top_data + i*dim);
  }
}

template <typename Dtype>
void NormalizeLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, vector<Blob<Dtype>*>* bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* top_data = top[0]->gpu_data();
  const Dtype* bottom_data = (*bottom)[0]->gpu_data();
  Dtype* bottom_diff = (*bottom)[0]->mutable_gpu_diff();
  int num = top[0]->num();
  int dim = top[0]->count() / num;
  Dtype a;
  for (int i = 0; i < num; ++i) {
    caffe_gpu_dot(dim, top_data + i*dim, top_diff + i*dim, &a);
    caffe_gpu_scale(dim, a, top_data + i*dim, bottom_diff + i*dim);
    caffe_gpu_sub(dim, top_diff + i*dim,
                    bottom_diff + i*dim,
                    bottom_diff + i*dim);
    /* skip the scaling to avoid diminishing gradients
    caffe_gpu_dot(dim, bottom_data + i*dim, bottom_data + i*dim, &a);
    caffe_gpu_scale(dim, Dtype(pow(a, -0.5)),
                    bottom_diff + i*dim,
                    bottom_diff + i*dim);
    */
  }
}

INSTANTIATE_CLASS(NormalizeLayer);


}  // namespace caffe
